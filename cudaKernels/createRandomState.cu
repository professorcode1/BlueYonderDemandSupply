#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

extern "C"{
    __global__ void initkernel(hiprandState_t* rndm_nmbr_gnrtr){
        int tidx = threadIdx.x + blockIdx.x * blockDim.x;
        hiprandState_t* s = new hiprandState_t;
        hiprand_init( %(SEED)s, tidx, 0, s);
        rndm_nmbr_gnrtr[tidx] = *s;
    }

}