/***
Assumptopn
    Invoked via (number of active grids,1,1) (bx,by,xz)
    where bx * by * xz = number of active threads 
***/
#define tIdx threadIdx.x
#define tIdy threadIdx.y
#define tIdz threadIdx.z

#define bDmx blockDim.x
#define bDmy blockDim.y
#define bDmz blockDim.z

#define bIdx blockIdx.x
#define bIdy blockIdx.y
#define bIdz blockIdx.z

#define gDmx gridDim.x
#define gDmy gridDim.y
#define gDmz gridDim.z

#define threadsPerBlock bDmx*bDmy*bDmz

#define WARP_SIZE 32


#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


__device__ __forceinline__ int row_major_4D(int w, int x, int y, int z, int W, int X,int Y, int Z){
    return z + Z * ( y + Y * ( x + X * w ) );
}

__device__ __forceinline__ int col_major_4D(int w, int x, int y, int z, int W, int X,int Y, int Z){
    return w + W *( x + X * (y + Y * z) );
}

__device__ __forceinline__ int col_major_3D(int x, int y, int z, int X,int Y, int Z){
    return x + X * (y + Y * z) ;
}

extern "C"{
__global__ void create_initial_population(int *population,const int population_size,const int time_frame,const int number_of_trucks,const int truck_capacity, 
    const int number_of_stores,const int number_of_products,const float probability_of_no_truck, hiprandState_t* rndm_nmbr_gnrtr){
        __shared__ int population_iter;
        __shared__ int time_iter;
        __shared__ int trucks_iter;
        __shared__ int population_per_block;
        __shared__ int block_solution_padding;
        
        int block_thread_id = col_major_3D(tIdx, tIdy, tIdz, bDmx, bDmy, bDmz);
        int global_thread_id = threadsPerBlock * bIdx + block_thread_id;
        
        if(block_thread_id == 0){
            population_per_block = population_size / gDmx + (population_size % gDmx != 0);
            block_solution_padding = population_per_block * bIdx;
            population_iter = 0;
        }
        __threadfence_block();
        
        while( population_iter * bDmx < population_per_block ){
            
            int thread_sol_index = block_solution_padding + population_iter * bIdx + tIdx;
       
            if(block_thread_id == 0){
                time_iter = 0;
            }
            __threadfence_block();

            while( time_iter * bDmy < time_frame ){
            
                int thread_time_index = time_iter * bIdy + tIdy;
                
                if(block_thread_id == 0){
                    trucks_iter = 0;
                }
                __threadfence_block();

                while( trucks_iter * bDmz < number_of_trucks ){
                    
                    int thread_truck_index = trucks_iter * bDmz + tIdz;

                    float rndm_nmbr_1, rndm_nmbr_2;
                    rndm_nmbr_1 = hiprand_uniform( rndm_nmbr_gnrtr + global_thread_id ) ;
                    rndm_nmbr_2 = hiprand_uniform( rndm_nmbr_gnrtr + global_thread_id ) ;
                    
                    int store = ceil(rndm_nmbr_1 * number_of_stores)  - 1; //curand_uniform includes 1 and not 0.
                    store = rndm_nmbr_2 <= probability_of_no_truck ? -1 : store;
                    int index = row_major_4D(thread_sol_index, thread_time_index, thread_truck_index, 0, population_size, time_frame,  number_of_trucks, truck_capacity);
                    population[ index ] = store; 

                    for(int thread_capacity_index = 1 ; thread_capacity_index <= truck_capacity ; thread_capacity_index++){
                        population[ index + thread_capacity_index ] = ceil(hiprand_uniform( rndm_nmbr_gnrtr + global_thread_id ) * number_of_products) - 1;
                    }
                    
                    if(block_thread_id == 0){
                        trucks_iter++;
                    }
                    __syncthreads();
                }                

                if(block_thread_id == 0){
                    time_iter++;
                }
                __syncthreads();
            
            }

            if( block_thread_id == 0 ){
                population_iter++;
            }
            __syncthreads();
        
        }
    }
}